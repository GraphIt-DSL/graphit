
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime_api.h>

int main(int argc, char *argv[]) {
    hipDeviceProp_t prop;
    hipError_t status;
    int device_count;
    int device_index = 0;
    if (argc > 1) {
        device_index = atoi(argv[1]);
    }

    status = hipGetDeviceCount(&device_count);
    if (status != hipSuccess) {
        fprintf(stderr,"cudaGetDeviceCount() failed: %s\n", hipGetErrorString(status));
        return -1;
    }
    if (device_index >= device_count) {
        fprintf(stderr, "Specified device index %d exceeds the maximum (the device count on this system is %d)\n", device_index, device_count);
        return -1;
    }
    status = hipGetDeviceProperties(&prop, device_index);
    if (status != hipSuccess) {
        fprintf(stderr,"cudaGetDeviceProperties() for device device_index failed: %s\n", hipGetErrorString(status));
        return -1;
    }
    int v = prop.major * 10 + prop.minor;
    printf("%d\n", v);
    printf("%d\n", prop.multiProcessorCount);
}
